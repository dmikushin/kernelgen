/*
 * Copyright (c) 2012 by Dmitry Mikushin
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) 
{
	// Create device context.
	hipDevice_t device;
	hipCtx_t context;
	hipError_t cuerr = hipInit(0);
	assert(cuerr == hipSuccess);
	cuerr = hipDeviceGet(&device, 0);
	assert(cuerr == hipSuccess);
	cuerr = hipCtxCreate(&context, hipDeviceScheduleSpin, device);
	assert(cuerr == hipSuccess);

	// Create an output value buffer.
	int* value;
	cuerr = hipMalloc((hipDeviceptr_t*)&value, sizeof(int));
	assert(cuerr == hipSuccess);

	// Load module.
	hipModule_t module;
	cuerr = hipModuleLoad(&module, "constant3.cubin");
	assert(cuerr == hipSuccess);

	// Load the unnamed constant (that is given a name, anyway).
	hipDeviceptr_t unnamed;
	size_t szunnamed = 0;
	cuerr = hipModuleGetGlobal(&unnamed, &szunnamed, module,
		"unnamedConst2_0");
	assert(cuerr == hipSuccess);
	printf("unnamedConst2_0 addr = %p size = %zu\n", (void*)unnamed, szunnamed);

	// Load the named constant.
	hipDeviceptr_t named;
	size_t sznamed = 0;
	cuerr = hipModuleGetGlobal(&named, &sznamed, module,
		"named");
	assert(cuerr == hipSuccess);
	printf("named addr = %p, size = %zu\n", (void*)named, sznamed);

	// Load kernel.
	hipFunction_t kernel;
	cuerr = hipModuleGetFunction(&kernel, module, "kernel");
	assert(cuerr == hipSuccess);
	
	// Configure kernel launch with output buffer parameter.
	cuerr = cuParamSetSize(kernel, 8);
	assert(cuerr == hipSuccess);
	cuerr = cuParamSetv(kernel, 0, &value, 8);
	assert(cuerr == hipSuccess);
	cuerr = cuFuncSetBlockShape(kernel, 1, 1, 1);
	assert(cuerr == hipSuccess);

	// Launch kernel.
	cuerr = cuLaunch(kernel);
	assert(cuerr == hipSuccess);

	// Wait for kernel completion.
	cuerr = hipCtxSynchronize();
	assert(cuerr == hipSuccess);

	// Get result from device memory.
	int cpu_value;
	cuerr = hipMemcpyDtoH(&cpu_value, (hipDeviceptr_t)value, sizeof(int));
	assert(cuerr == hipSuccess);
	printf("initial result = %d\n", cpu_value);

	// Change the value in contant memory and run
	// kernel again to copy it to the output buffer.
	cpu_value = 10;
	cuerr = hipMemcpyHtoD(unnamed, &cpu_value, sizeof(int));
	assert(cuerr == hipSuccess);
	cpu_value = 0;

	// Configure kernel launch with output buffer parameter.
	cuerr = cuParamSetSize(kernel, 8);
	assert(cuerr == hipSuccess);
	cuerr = cuParamSetv(kernel, 0, &value, 8);
	assert(cuerr == hipSuccess);
	cuerr = cuFuncSetBlockShape(kernel, 1, 1, 1);
	assert(cuerr == hipSuccess);

	// Launch kernel.
	cuerr = cuLaunch(kernel);
	assert(cuerr == hipSuccess);

	// Wait for kernel completion.
	cuerr = hipCtxSynchronize();
	assert(cuerr == hipSuccess);

	// Get result from device memory.
	cuerr = hipMemcpyDtoH(&cpu_value, (hipDeviceptr_t)value, sizeof(int));
	assert(cuerr == hipSuccess);
	printf("changed result = %d\n", cpu_value);

	// Free output buffer.
	cuerr = hipFree((hipDeviceptr_t)value);
	assert(cuerr == hipSuccess);

	// Unload module and destroy context.
        cuerr = hipModuleUnload(module);
	assert(cuerr == hipSuccess);
	cuerr = hipCtxDestroy(context);
	assert(cuerr == hipSuccess);

        return 0;
}

