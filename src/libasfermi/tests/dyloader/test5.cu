#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012 by Dmitry Mikushin
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */

#include "hip/hip_runtime.h"
#include "cuda_dyloader.h"

__global__ void kernel1(int* lock)
{
	// Wait for unlock.
	while (atomicCAS(lock, 0, 0)) continue;
}

/*extern "C" __global__ void kernel2(int* lock)
{
	// Unlock.
	atomicCAS(lock, 1, 0);
}*/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

static void usage(const char* filename)
{
	printf("Usage: %s <mode>\n", filename);
	printf("mode = 0: launch kernel1 and then load kernel2 with hipModuleGetFunction (will hang)\n");
	printf("mode = 1: launch kernel1 and then load kernel2 using dyloader (will succeed)\n");
}

int main (int argc, char* argv[])
{
	if (argc != 2)
	{
		usage(argv[0]);
		return 0;
	}

	int mode = atoi(argv[1]);
	if ((mode < 0) || (mode > 2))
	{
		usage(argv[0]);
		return 0;
	}

	// Initialize lock.
	int* lock = NULL;
	hipError_t cuerr = hipMalloc((void**)&lock, sizeof(int));
	assert(cuerr == hipSuccess);
	int one = 1;
	cuerr = hipMemcpy(lock, &one, sizeof(int), hipMemcpyHostToDevice);
	assert(cuerr == hipSuccess);

	// Dynamic loader initialization is synchronous,
	// no way.
	CUDYloader loader;
	if (mode == 1)
	{
		// Space for dynamically loaded kernels.
		int capacity = 100;
	
		// Initialize dynamic loader.
		hipError_t err = cudyInit(&loader, capacity);
		assert(err == hipSuccess);
	}

	// Create streams.
	hipStream_t stream1, stream2;
	cuerr = hipStreamCreate(&stream1);
	assert(cuerr == hipSuccess);
	cuerr = hipStreamCreate(&stream2);
	assert(cuerr == hipSuccess);

	// Launch first kernel.
	kernel1<<<1, 1, 0, stream1>>>(lock);
	cuerr = hipGetLastError();
	assert(cuerr == hipSuccess);

	printf("Submitted kernel1\n");

	if (mode == 0)
	{
		// Load second kernel.
		hipModule_t module;
		hipError_t err = hipModuleLoad(&module, "kernel2.cubin");
		assert(err == hipSuccess);
		hipFunction_t kernel2;
		err = hipModuleGetFunction(&kernel2, module, "kernel2");

		struct { unsigned int x, y, z; } gridDim, blockDim;
		gridDim.x = 1; gridDim.y = 1; gridDim.z = 1;
		blockDim.x = 1; blockDim.y = 1; blockDim.z = 1;
		size_t szshmem = 0;
		void* kernel2_args[] = { (void*)&lock };
		err = hipModuleLaunchKernel(kernel2,
			gridDim.x, gridDim.y, gridDim.z,
			blockDim.x, blockDim.y, blockDim.z, szshmem,
			stream2, kernel2_args, NULL);
		assert(err == hipSuccess);
		
		printf("Sumbitted kernel2\n");

		cuerr = hipDeviceSynchronize();
		assert(cuerr == hipSuccess);
	}
	else
	{
		// Load kernel function from the binary opcodes.
		CUDYfunction function;
		hipError_t err = cudyLoadCubin(&function,
			loader, "kernel2.cubin", "kernel2", stream2);
		assert(err == hipSuccess);

		// Launch kernel function within dynamic loader.
		err = cudyLaunch(function,
			1, 1, 1, 1, 1, 1, 0, &lock, stream2);
		assert(err == hipSuccess);
		
		printf("Submitted kernel2\n");

		cuerr = hipDeviceSynchronize();
		assert(cuerr == hipSuccess);

		err = cudyDispose(loader);
		assert(err == hipSuccess);
	}

	printf("Finished\n");

	cuerr = hipStreamDestroy(stream1);
	assert(cuerr == hipSuccess);
	cuerr = hipStreamDestroy(stream2);
	assert(cuerr == hipSuccess);

	cuerr = hipFree(lock);
	assert(cuerr == hipSuccess);

	return 0;
}

