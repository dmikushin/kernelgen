#include "hip/hip_runtime.h"
#include "kernelgen_memory.h"

#include <stdio.h>

// Setup the device global memory pool initial configuration.
void kernelgen_memory_init(size_t szpool)
{
	// First, fill config on host.
	kernelgen_memory_t config_host;

	// Allocate pool and flush it to zero.
	hipError_t cuerr = hipMalloc((void**)&config_host.pool, szpool);
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot allocate device memory pool: %s\n",
			hipGetErrorString(cuerr));
		return;
	}
	cuerr = hipMemset(config_host.pool, 0, szpool);
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot initialize device memory pool: %s\n",
			hipGetErrorString(cuerr));
		return;
	}

	config_host.szused = 0;
	config_host.szpool = szpool;
	config_host.count = 0;

	// Copy the resulting config to the special
	// device variable.
	kernelgen_memory_t* config_device;
	cuerr = hipGetSymbolAddress((void**)&config_device, "kernelgen_memory");
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot find kernelgen_memory on device: %s\n",
			hipGetErrorString(cuerr));
		return;
	}
	cuerr = hipMemcpy(config_device, &config_host,
		sizeof(kernelgen_memory_t), hipMemcpyHostToDevice);
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot copy device memory pool configuration: %s\n",
			hipGetErrorString(cuerr));
		return;
	}
}

__global__ void kernel()
{
	char* A = (char*)kernelgen_malloc(1);
	A[0] = 'A';
	char* B = (char*)kernelgen_malloc(2);
	B[1] = 'B';
	char* C = (char*)kernelgen_malloc(3);
	C[1] = 'C';
	kernelgen_free(B);
	char* D = (char*)kernelgen_malloc(1);
	D[0] = 'D';
}

int main(int argc, char* argv[])
{
	size_t szpool = 80;
	kernelgen_memory_init(szpool);

	kernel<<<1,1>>>();
	hipError_t cuerr = hipDeviceSynchronize();
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot synchronize test kernel: %s\n",
			hipGetErrorString(cuerr));
		return -1;
	}

	// Check the pool contents.
	kernelgen_memory_t config_host, *config_device;
	cuerr = hipGetSymbolAddress((void**)&config_device, "kernelgen_memory");
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot find kernelgen_memory on device: %s\n",
			hipGetErrorString(cuerr));
		return -1;
	}
	cuerr = hipMemcpy(&config_host, config_device,
		sizeof(kernelgen_memory_t), hipMemcpyDeviceToHost);
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot copy back device memory pool configuration: %s\n",
			hipGetErrorString(cuerr));
		return -1;
	}
	char* pool = (char*)malloc(szpool);
	cuerr = hipMemcpy(pool, config_host.pool, szpool, hipMemcpyDeviceToHost);
	for (int i = 0; i < szpool; i++)
		if ((pool[i] >= 'A') && (pool[i] <= 'Z'))
			printf("%c", pool[i]);
		else
			printf(" ");
	printf("\n");
	for (int i = 0; i < szpool; i++)
		printf("%d", i % 8);
	printf("\n");

	return 0;
}

