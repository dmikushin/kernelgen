__device__ int spinval;
__device__ void spin(int count)
{
	volatile int *spinptr = &spinval;
	while(count--)
	{
		*spinptr = count;

		// ~500 cycles timeout on Fermi
		__threadfence();
	}
}

__global__ void kernel1(int* lock)
{
	// Wait for unlock.
	while (atomicCAS(lock, 0, 0))
		spin(10);
}

__global__ void kernel2(int* lock)
{
	// Unlock.
	atomicCAS(lock, 1, 0);
}

__global__ void kernel3(int* lock)
{
	// Device-malloc call.
	int** buffer = (int**)malloc(10);
	buffer[1] = lock;

	// Unlock.
	atomicCAS(lock, 1, 0);
}

#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

static void usage(const char* filename)
{
	printf("Usage: %s <mode>\n", filename);
	printf("mode = 0: launch kernel1 and kernel2 without device-malloc (will succeed)\n");
	printf("mode = 1: launch kernel1 and kernel3 with device-malloc (will hang)\n");
}

int main (int argc, char* argv[])
{
	if (argc != 2)
	{
		usage(argv[0]);
		return 0;
	}

	int mode = atoi(argv[1]);
	if ((mode < 0) || (mode > 1))
	{
		usage(argv[0]);
		return 0;
	}

	// Initialize lock.
	int* lock = NULL;
	hipError_t cuerr = hipMalloc((void**)&lock, sizeof(int));
	assert(cuerr == hipSuccess);
	int one = 1;
	cuerr = hipMemcpy(lock, &one, sizeof(int), hipMemcpyHostToDevice);
	assert(cuerr == hipSuccess);

	// Create streams.
	hipStream_t stream1, stream2;
	cuerr = hipStreamCreate(&stream1);
	assert(cuerr == hipSuccess);
	cuerr = hipStreamCreate(&stream2);
	assert(cuerr == hipSuccess);

	if (mode == 0)
	{
		// Launch first kernel.
		kernel1<<<1, 1, 0, stream1>>>(lock);
		cuerr = hipGetLastError();
		assert(cuerr == hipSuccess);

		// Launch second kernel.
		kernel2<<<1, 1, 0, stream2>>>(lock);
		cuerr = hipGetLastError();
		assert(cuerr == hipSuccess);
	}
	if (mode == 1)
	{
		// Launch first kernel.
		kernel1<<<1, 1, 0, stream1>>>(lock);
		cuerr = hipGetLastError();
		assert(cuerr == hipSuccess);

		// Launch third kernel (with malloc).
		kernel3<<<1, 1, 0, stream2>>>(lock);
		cuerr = hipGetLastError();
		assert(cuerr == hipSuccess);
	}

	cuerr = hipDeviceSynchronize();
	assert(cuerr == hipSuccess);

	cuerr = hipStreamDestroy(stream1);
	assert(cuerr == hipSuccess);
	cuerr = hipStreamDestroy(stream2);
	assert(cuerr == hipSuccess);

	cuerr = hipFree(lock);
	assert(cuerr == hipSuccess);

	return 0;
}
