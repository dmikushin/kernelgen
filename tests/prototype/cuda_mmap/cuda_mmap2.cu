#include <hip/hip_runtime.h>
#include <list>
#include <signal.h>
#include <stdio.h>
#include <sys/mman.h>
#include <sys/types.h>
#include <unistd.h>

std::list<void*> maps;

// signal handler
void sighandler(int code, siginfo_t *siginfo, void* ucontext)
{
	// Check if address is valid on GPU.
	void* addr = siginfo->si_addr;

	void* base;
	size_t size;
	hipError_t cuerr = hipMemGetAddressRange((hipDeviceptr_t*)&base, &size, (hipDeviceptr_t)addr);
	if (cuerr == hipSuccess)
	{
		void* map = mmap(base, size,
			PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS,
			-1, 0);
		if (map == (void*)-1)
		{
			fprintf(stderr, "Cannot map memory onto %p + %zu\n",
				base, size);
			return;
		}
		maps.push_back(map);

		printf("Mapped memory %p + %zu onto %p + %zu\n",
			map, size, base, size);

		hipError_t cuerr = hipMemcpy(base, base, size, hipMemcpyDeviceToHost);
		if (cuerr != hipSuccess)
		{
			fprintf(stderr, "Cannot copy GPU data to mapped memory: %s\n",
				hipGetErrorString(cuerr));
			return;
		}
	}
}

__global__ void kernel(int* array)
{
	array[10] = 1313;
}

int main(int argc, char* argv[])
{
	// Set up signal handler.
	struct sigaction sa;
	sa.sa_flags = SA_SIGINFO;
	sigfillset(&sa.sa_mask);
	sa.sa_sigaction = sighandler;
	sigaction(SIGSEGV, &sa, 0);

	// Create GPU array.
	size_t length = 103 * sizeof(int);
	int* array = NULL;
	hipError_t cuerr = hipMalloc((void**)&array, length);
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot create device memory array: %s\n",
			hipGetErrorString(cuerr));
		return -1;
	}

	// Launch GPU kernel, assigning value to GPU array.
	kernel<<<1, 1>>>(array);
	cuerr = hipDeviceSynchronize();
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot synchronize GPU kernel: %s\n",
			hipGetErrorString(cuerr));
		return -1;
	}

	printf("Trying to read: array[10] = %d\n", array[10]);

	for (std::list<void*>::iterator i = maps.begin(), e = maps.end(); i != e; i++)
	{
		void* map = *i;
		int err = munmap(map, length);
		if (err == -1)
		{
			fprintf(stderr, "Cannot unmap memory from %p + %zu\n",
				array, length);
			return -1;
		}
	}

	return 0;
}

