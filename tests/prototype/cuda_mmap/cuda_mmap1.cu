
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/mman.h>

__global__ void kernel(int* array)
{
	array[10] = 1313;
}

int main(int argc, char* argv[])
{
	size_t length = 103 * sizeof(int);
	int* array = NULL;
	hipError_t cuerr = hipMalloc((void**)&array, length);
	kernel<<<1, 1>>>(array);
	cuerr = hipDeviceSynchronize();

	void* map = mmap(array, length,
		PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS,
		-1, 0);
	if (map == (void*)-1)
	{
		fprintf(stderr, "Cannot map memory onto %p + %zu\n",
			array, length);
		return -1;
	}

	printf("Mapped memory %p + %zu onto %p + %zu\n",
		map, length, array, length);
	printf("Before assignment: array[10] = %d\n", array[10]);

	cuerr = hipMemcpy(array, array, length, hipMemcpyDeviceToHost);
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot copy GPU data to mapped memory: %s\n",
			hipGetErrorString(cuerr));
		return 1;
	}

	printf("After assignment: array[10] = %d\n", array[10]);

	int err = munmap(map, length);
	if (err == -1)
	{
		fprintf(stderr, "Cannot unmap memory from %p + %zu\n",
			array, length);
		return -1;
	}

	return 0;
}

