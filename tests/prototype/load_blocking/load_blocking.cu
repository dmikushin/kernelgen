__device__ int spinval;
__device__ void spin(int count)
{
	volatile int *spinptr = &spinval;
	while(count--)
	{
		*spinptr = count;

		// ~500 cycles timeout on Fermi
		__threadfence();
	}
}

__global__ void kernel1(int* lock)
{
	// Wait for unlock.
	while (atomicCAS(lock, 0, 0)) spin(10);
}

/*extern "C" __global__ void kernel2(int* lock)
{
	// Unlock.
	atomicCAS(lock, 1, 0);
}*/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

static void usage(const char* filename)
{
	printf("Usage: %s <mode>\n", filename);
	printf("mode = 0: launch kernel1 before kernel2 load (will hang)\n");
	printf("mode = 1: launch kernel1 after kernel2 load (will succeed)\n");
}

int main (int argc, char* argv[])
{
	if (argc != 2)
	{
		usage(argv[0]);
		return 0;
	}

	int mode = atoi(argv[1]);
	if ((mode < 0) || (mode > 2))
	{
		usage(argv[0]);
		return 0;
	}

	// Initialize lock.
	int* lock = NULL;
	hipError_t cuerr = hipMalloc((void**)&lock, sizeof(int));
	assert(cuerr == hipSuccess);
	int one = 1;
	cuerr = hipMemcpy(lock, &one, sizeof(int), hipMemcpyHostToDevice);
	assert(cuerr == hipSuccess);

	// Create streams.
	hipStream_t stream1, stream2;
	cuerr = hipStreamCreate(&stream1);
	assert(cuerr == hipSuccess);
	cuerr = hipStreamCreate(&stream2);
	assert(cuerr == hipSuccess);

	if (mode == 0)
	{
		// Launch first kernel.
		kernel1<<<1, 1, 0, stream1>>>(lock);
		cuerr = hipGetLastError();
		assert(cuerr == hipSuccess);
	}

	// Load second kernel.
	hipModule_t module;
	hipError_t err = hipModuleLoad(&module, "kernel2.ptx");
	assert(err == hipSuccess);
	hipFunction_t kernel2;
	err = hipModuleGetFunction(&kernel2, module, "kernel2");

	if (mode == 1)
	{
		// Launch first kernel.
		kernel1<<<1, 1, 0, stream1>>>(lock);
		cuerr = hipGetLastError();
		assert(cuerr == hipSuccess);
	}

	struct { unsigned int x, y, z; } gridDim, blockDim;
	gridDim.x = 1; gridDim.y = 1; gridDim.z = 1;
	blockDim.x = 1; blockDim.y = 1; blockDim.z = 1;
	size_t szshmem = 0;
	void* kernel2_args[] = { (void*)&lock };
	err = hipModuleLaunchKernel(kernel2,
		gridDim.x, gridDim.y, gridDim.z,
		blockDim.x, blockDim.y, blockDim.z, szshmem,
		stream2, kernel2_args, NULL);
	assert(err == hipSuccess);

	cuerr = hipDeviceSynchronize();
	assert(cuerr == hipSuccess);

	cuerr = hipStreamDestroy(stream1);
	assert(cuerr == hipSuccess);
	cuerr = hipStreamDestroy(stream2);
	assert(cuerr == hipSuccess);

	cuerr = hipFree(lock);
	assert(cuerr == hipSuccess);

	return 0;
}
